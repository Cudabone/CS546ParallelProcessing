/* Matrix normalization.
 * Compile with "gcc matrixNorm.c" 
 */

/* ****** ADD YOUR CODE AT THE END OF THIS FILE. ******
 * You need not submit the provided code.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
volatile float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
void matrixNorm();
void cudaErrorCheck(hipError_t err, const char *s);
__global__ void matrixCuda(float *d_A,int N);
__global__ void BlockMean(float *d_A,float *d_Sum, int N);
__global__ void BlockDev(float *d_A, float *d_Dev, float *d_mu,int N);
__global__ void Normalize(float *d_A, float *d_mu, float *d_sigma, int N);

/* returns a seed for srand based on the time */
unsigned int time_seed() {
	struct timeval t;
	struct timezone tzdummy;

	gettimeofday(&t, &tzdummy);
	return (unsigned int)(t.tv_usec);
}

/* Set the program parameters from the command-line arguments */
void parameters(int argc, char **argv) {
	int seed = 0;  /* Random seed */
	char uid[32]; /*User name */

	/* Read command-line arguments */
	srand(time_seed());  /* Randomize */

	if (argc == 3) {
		seed = atoi(argv[2]);
		srand(seed);
		printf("Random seed = %i\n", seed);
	} 
	if (argc >= 2) {
		N = atoi(argv[1]);
		if (N < 1 || N > MAXN) {
			printf("N = %i is out of range.\n", N);
			exit(0);
		}
	}
	else {
		printf("Usage: %s <matrix_dimension> [random seed]\n",
				argv[0]);    
		exit(0);
	}

	/* Print parameters */
	printf("\nMatrix dimension N = %i.\n", N);
}

/* Initialize A and B*/
void initialize_inputs() {
	int row, col;

	printf("\nInitializing...\n");

	for (col = 0; col < N; col++) {
		for (row = 0; row < N; row++) {
			A[row][col] = (float)rand() / 32768.0;
			B[row][col] = 0.0;
		}
	}
	/*
	   for (col = 0; col < N; col++) {
	   for (row = 0; row < N; row++) {
	   A[row][col] = col + row;
	   B[row][col] = 0.0;
	   }
	   }
	   */

}

/* Print input matrices */
void print_inputs() {
	int row, col;

	if (N < 10) {
		printf("\nA =\n\t");
		for (row = 0; row < N; row++) {
			for (col = 0; col < N; col++) {
				printf("%5.2f%s", A[row][col], (col < N-1) ? ", " : ";\n\t");
			}
		}
	}
}

void print_B() {
	int row, col;

	if (N < 10) {
		printf("\nB =\n\t");
		for (row = 0; row < N; row++) {
			for (col = 0; col < N; col++) {
				printf("%1.10f%s", B[row][col], (col < N-1) ? ", " : ";\n\t");
			}
		}
	}
}

int main(int argc, char **argv) {
	/* Timing variables */
	struct timeval etstart, etstop;  /* Elapsed times using gettimeofday() */
	struct timezone tzdummy;
	clock_t etstart2, etstop2;  /* Elapsed times using times() */
	unsigned long long usecstart, usecstop;
	struct tms cputstart, cputstop;  /* CPU times for my processes */

	/* Process program parameters */
	parameters(argc, argv);

	/* Initialize A and B */
	initialize_inputs();

	/* Print input matrices */
	print_inputs();

	/* Start Clock */
	printf("\nStarting clock.\n");
	gettimeofday(&etstart, &tzdummy);
	etstart2 = times(&cputstart);

	/* Gaussian Elimination */
	matrixNorm();

	/* Stop Clock */
	gettimeofday(&etstop, &tzdummy);
	etstop2 = times(&cputstop);
	printf("Stopped clock.\n");
	usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
	usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

	/* Display output */
	print_B();

	/* Display timing results */
	printf("\nElapsed time = %g ms.\n",
			(float)(usecstop - usecstart)/(float)1000);

	printf("(CPU times are accurate to the nearest %g ms)\n",
			1.0/(float)CLOCKS_PER_SEC * 1000.0);
	printf("My total CPU time for parent = %g ms.\n",
			(float)( (cputstop.tms_utime + cputstop.tms_stime) -
				(cputstart.tms_utime + cputstart.tms_stime) ) /
			(float)CLOCKS_PER_SEC * 1000);
	printf("My system CPU time for parent = %g ms.\n",
			(float)(cputstop.tms_stime - cputstart.tms_stime) /
			(float)CLOCKS_PER_SEC * 1000);
	printf("My total CPU time for child processes = %g ms.\n",
			(float)( (cputstop.tms_cutime + cputstop.tms_cstime) -
				(cputstart.tms_cutime + cputstart.tms_cstime) ) /
			(float)CLOCKS_PER_SEC * 1000);
	/* Contrary to the man pages, this appears not to include the parent */
	printf("--------------------------------------------\n");

	exit(0);
}

/* ------------------ Above Was Provided --------------------- */

/****** You will replace this routine with your own parallel version *******/
/* Provided global variables are MAXN, N, A[][] and B[][],
 * defined in the beginning of this code.  B[][] is initialized to zeros.
 */


/* Matrix Normalization for the Cuda platform
   Overview
   1.Copy matrix A to Device
   2.CUDA: Split matrix into 2D grid of blocks, each block calculates a partial sum 
   for each column in their section via BlockMean.
   3.Sequentially add all of these partial sums to form the total mean for
   each column.
   4.CUDA: Each block calculate a partial sum of the mean difference for each
   column in their section via BlockDev.
   5.Sequentially add all of these partial mean differences to form the total
   standard deviation for each column.
   6.CUDA: Each block normalize their portion of the matrix using the means
   and standard deviations calculated.
   7.Copy matrix A to Host's B
   */

//Number of threads per block
#define BlockSize 32
void matrixNorm() 
{
	printf("Executing on GPU\n");

	//Set grid size to divide among number of threads
	int GridSize = ceil((float)N/BlockSize); 

	//Create CUDA grid and block size for matrix
	dim3 grid(GridSize,GridSize);
	dim3 block(BlockSize,BlockSize);

	//Device matrices and vectors for calculation 
	float *d_A; //the matrix 
	float *d_Sum; //a partial sum holder
	float *d_Dev; //a partial deviation holder
	float *d_mu; //a vector of means for each column
	float *d_sigma; //a vector of standard deviations for each column

	//Host copies of variables to initalize the device's copies to 0's
	float *h_Sum = (float *)malloc(GridSize*N*sizeof(float));
	float *h_Dev = (float *)malloc(GridSize*N*sizeof(float));
	float *h_mu = (float *)malloc(N*sizeof(float));
	float *h_sigma = (float *)malloc(N*sizeof(float));
	size_t size = N*N*sizeof(float);

	// Allocate Matrices on Devices 
	cudaErrorCheck(hipMalloc((void **)&d_A,size), "hipMalloc A");
	cudaErrorCheck(hipMalloc(&d_Sum,GridSize*N*sizeof(float)), "hipMalloc d_Sum" );
	cudaErrorCheck(hipMalloc(&d_Dev,GridSize*N*sizeof(float)), "hipMalloc d_Dev ");
	cudaErrorCheck(hipMalloc((void **)&d_mu,N*sizeof(float)), "hipMalloc d_mu");
	cudaErrorCheck(hipMalloc((void **)&d_sigma,N*sizeof(float)), "hipMalloc d_sigma");

	// Copy over matrix to device 
	cudaErrorCheck(hipMemcpy(d_A,(const void
					*)A[0],size,hipMemcpyHostToDevice), "hipMemcpy A");

	//Initalize h_sum and h_dev to 0
	int row,col;
	for(row = 0; row < GridSize; row++)
	{
		h_mu[row] = 0.0;
		h_sigma[row] = 0.0;
		for(col = 0; col < N; col++)
		{
			h_Sum[row*N + col] = 0.0;
			h_Dev[row*N + col] = 0.0;
		}
	}

	//Initalize the device sum and std dev arrays to 0s.
	cudaErrorCheck(hipMemcpy((void *)d_Sum,(const void
				*)h_Sum,GridSize*N*sizeof(float),hipMemcpyHostToDevice),
			"hipMemcpy to d_Sum");
	cudaErrorCheck(hipMemcpy((void *)d_Dev,(const void
				*)h_Dev,GridSize*N*sizeof(float),hipMemcpyHostToDevice),"hipMemcpy to d_Dev" );
	cudaErrorCheck(hipMemcpy((void *)d_mu,(const void
				*)h_mu,N*sizeof(float),hipMemcpyHostToDevice), "hipMemcpy to d_mu" );
	cudaErrorCheck(hipMemcpy((void *)d_sigma,(const void
				*)h_sigma,N*sizeof(float),hipMemcpyHostToDevice), "hipMemcpy to d_sigma");


	//Calcuate a sub mean for each block
	BlockMean<<<grid,block>>>(d_A,d_Sum,N);
	hipDeviceSynchronize();

	//Calculate total mean for each column sequentially 
	cudaErrorCheck(hipMemcpy((void *)h_Sum,(const void
					*)d_Sum,GridSize*N*sizeof(float),hipMemcpyDeviceToHost),
			"hipMemcpy to h_Sum");

	for(row = 0; row < GridSize; row++)
	{
		for(col = 0; col < N; col++)
		{
			h_mu[col] += h_Sum[row*N + col];
		}
	} 
	for(col = 0; col < N; col++)
		h_mu[col] /= N; 

	//Copy over host calculated mu vector to host
	cudaErrorCheck(hipMemcpy((void *)d_mu,(const void
				*)h_mu,N*sizeof(float),hipMemcpyHostToDevice), "hipMemcpy to d_mu");

	//Calculate a sub standard deviation for each block
	BlockDev<<<grid,block>>>(d_A,d_Dev,d_mu,N);
	hipDeviceSynchronize();

	//Calculate total standard deviation from each block sequentially
	cudaErrorCheck(hipMemcpy((void *)h_Dev,(const void
				*)d_Dev,GridSize*N*sizeof(float),hipMemcpyDeviceToHost),
			"hipMemcpy to h_dev");

	for(row = 0; row < GridSize; row++)
	{
		for(col = 0; col < N; col++)
		{
			h_sigma[col] += h_Dev[row*N + col];
		}
	}
	for(col = 0; col < N; col++)
	{
		h_sigma[col] /= N;
		h_sigma[col] = sqrt(h_sigma[col]);
	}

	//Copy sigma vector to device
	cudaErrorCheck(hipMemcpy((void *)d_sigma,(const void
				*)h_sigma,N*sizeof(float),hipMemcpyHostToDevice), "hipMemcpy to d_sigma" );

	//Normalize with means and standard deviations by splitting into blocks
	Normalize<<<grid,block>>>(d_A,d_mu,d_sigma,N);
	hipDeviceSynchronize();

	//Copy Normalized array back to B
	cudaErrorCheck(hipMemcpy((void *)B[0],(const void
					*)d_A,size,hipMemcpyDeviceToHost), "hipMemcpy to B");

	//Free all host and device pointers
	hipFree(d_A);
	hipFree(d_Sum);
	hipFree(d_Dev);
	hipFree(d_mu);
	hipFree(d_sigma);
	free(h_Sum);
	free(h_Dev);
	free(h_mu);
	free(h_sigma);
}

/* Calculates a partial sum for a section of the matrix for each block.
   This is done by allocating a shared sub matrix, calculated a
   sum for each column using the algorithm from class, and returning
   the corresponding sub sum from each block.
   d_Sum holds all of these partial sums for every block.
   */
__global__ void BlockMean(float *d_A,float *d_Sum, int N)
{
	//Shared sub matrix
	__shared__ float sum[BlockSize*BlockSize];

	//Block and thread indices
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	//Size of entire matrix
	int size = N*N*sizeof(float);

	//Indices for flattened input matrices
	// i + j == [i][j]
	unsigned int i = blockIdx.x*BlockSize*N + tx*N;
	unsigned int j = blockIdx.y*BlockSize + ty;

	//Row index into sum
	unsigned int sx = tx*BlockSize;

	//Ensure block and thread within bounds of matrix
	if(x >= N || y >= N)
		return; 

	//Transfer section of d_A into sum
	if(i + j < size) 
		sum[sx + ty] = d_A[i + j];  
	else 
		sum[sx + ty] = 0.0;

	//Apply partial sum algorithm from class
	for(unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
	{
		__syncthreads();
		if(tx < stride)
			sum[sx + ty] += sum[sx + ty + stride];
	}
	
	//Transfer shared sub sum matrix to global memory
	if(tx == 0)
	{
		d_Sum[blockIdx.x*N + ty] = sum[ty];
	}
}

/* Calculate a partial sum of the square of the difference between the mean 
   for each block, in the same way as BlockMean, except by squaring a 
   difference of a calculated mean for the column in d_mu.
   */
__global__ void BlockDev(float *d_A, float *d_Dev, float *d_mu, int N)
{
	//Shared mu vector
	__shared__ float mu[BlockSize];
	//shared partial sum sub matrix
	__shared__ float sum[BlockSize*BlockSize];
	//Size of entire matrix
	int size = N*N*sizeof(float);

	//Block and thread indices
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	//Indices for flattened input matrices
	// i + j == [i][j]
	unsigned int i = blockIdx.x*BlockSize*N + tx*N;
	unsigned int j = blockIdx.y*BlockSize + ty;

	//Row index into sum
	unsigned int sx = tx*BlockSize;

	//Ensure block and thread within bounds of matrix
	if(x >= N || y >= N)
		return; 

	//Transfer sub mu vector into shared memory
	if(tx == 0)
	{
		mu[ty] = d_mu[j];
	}
	
	//Ensure mu vector populated
	__syncthreads();

	//Transfer section of d_A into sum and square the mean difference
	if((i + j < size) && (j < N))
		sum[sx + ty] = powf(d_A[i + j] - mu[ty],2.0);  
	else 
		sum[sx + ty] = 0.0;

	
	//Apply partial sum algorithm shown in class
	for(unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
	{
		__syncthreads();
		if(tx < stride)
			sum[sx + ty] += sum[sx + ty + stride];
	}

	//Tranfer shared sub matrix to global memory
	if(tx == 0)
	{
		d_Dev[blockIdx.x*N + ty] = sum[ty];
	}
}
/* Normalizing function, which each thread among all blocks corresponds
   to a single element in the matrix. Each applies the normalizing function
   to its element*/
__global__ void Normalize(float *d_A, float *d_mu, float *d_sigma, int N)
{
	// Thread position variables
	unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;

	//Ensure within bounds of matrix
	if(tx >= N || ty >= N)
		return;

	//If sigma 0, set 0, else calculate normalized value
	if(d_sigma[ty] == 0)
		d_A[tx + ty*N] = 0;
	else
		d_A[tx + ty*N] = (d_A[tx + ty*N] - d_mu[ty])/(d_sigma[ty]);
}

/*Simple printing error function */
void cudaErrorCheck(hipError_t err, const char *s)
{
	if(err != hipSuccess)
	{
		printf("%s error: %s\n",s,hipGetErrorString(err));
		exit(0);
	}
}

